#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/


#include "reference_calc.cpp"
#include "utils.h"
#include <math.h>


// function for calculating the min; use shared memory and reduce primitive 
__global__ void minmax_shmem_reduce_kernal(float *d_out, const float* d_in, const int NUMTOLPIXEL, bool min)
{  
  // the indexes; the "global" location in the whole image
  const int  pixel_1D_global = blockIdx.x * blockDim.x + threadIdx.x;

  // make sure the pixel being processed is indeed within the image
  if (pixel_1D_global >= NUMTOLPIXEL){
    return;
  }

  // the indexes; the location within the block, i.e, their thread indexes; used to index the sh_image
  const int pixel_1D_block = threadIdx.x;

  // The Shared memory is to store the image block
  extern __shared__ float sh_image[];
  sh_image[pixel_1D_block] = d_in[pixel_1D_global];
  __syncthreads(); // make sure entire block is loaded

  // do reduction in shared mem
  for (int s = blockDim.x / 2 ; s > 0; s >>= 1){
    if (pixel_1D_block < s){
      if(min){
		sh_image[pixel_1D_block] = min(sh_image[pixel_1D_block], sh_image[pixel_1D_block + s];
      }
      else
      {
		sh_image[pixel_1D_block] = max(sh_image[pixel_1D_block], sh_image[pixel_1D_block + s];
      }
    }
    __syncthreads(); // make sure all operations at one stage are done
  }

  // only thread 0 writes the result for this block back to globel mem
  if (pixel_1D_block == 0){
    d_out[blockIdx.x] = sh_image[0];
  }

}


// function to calculating calculate the hist gram
__global__ void histo(int * d_bins, const float* d_in, const size_t NUMTOLPIXEL, const size_t numBins, const float min_logLum, const float lumRange)
{

  // the indexes; the "global" location in the whole image
  const int  pixel_1D_global = blockIdx.x * blockDim.x + threadIdx.x;

  // make sure the pixel being processed is indeed within the image
  if (pixel_1D_global >= NUMTOLPIXEL){
    return;
  }

  float lum = d_in[pixel_1D_global];
  int   bin = (lum - min_logLum ) / lumRange * numBins;
  atomicAdd(&(d_bins[bin]), 1);
}

// function to calculate min or max using reduce



// function to do the exclusive scan
float reduce_minmax(const float* const d_logLuminance, const size_t NUMTOLPIXEL, bool minFlag)
{

	const int BLOCKSIZE =   32;
  
	int gridSize = ceil(NUMTOPIXEL / BLOCKSIZE);
  
	// declare intermediate GPU memory pointers
	float *d_out, *d_in;

	// allocate memory for them
	checkCudaErrors(hipMalloc( (void **) &d_in, sizeof(float) * size));    
	checkCudaErrors(hipMemcpy(d_in, d_logLuminance, sizeof(float) * size, hipMemcpyDeviceToDevice));
  
	checkCudaErrors( hipMalloc( (void **) &d_out,  sizeof(float) * gridSize ) );

	
	minmax_shmem_reduce_kernal<<<gridSize, BLOCKSIZE, sizeof(float) * BLOCKSIZE >>> (d_out, d_logLuminance, NUMTOLPIXEL, minFlag);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  
	// Keep calling the kernal until the gridSize reaches 1
	while(gridSize > 1 ){
		// first update input to be the output from pervious kernel call
		checkCudaErros(hipFree( d_in) );
		d_in = d_out;

		// update the gridSize
		gridSize = ceil( gridSize / BLOCKSIZE);

		// Call the kernal again
		minmax_shmem_reduce_kernal<<<gridSize, BLOCKSIZE, sizeof(float) * BLOCKSIZE >>> (d_out, d_in, NUMTOLPIXEL, minFlag);
		hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
	}

	float result = d_out[0];
	checkCudaErrors(CudaFree(d_out));
	return result;
}
  
void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum        
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */

  // ZL: Use 1D block and thread to simply the indexing through out the whole file
	const size_t NUMTOLPIXEL  = numRows * numCols;
	const int MAXTHREADPERBLOCK = 1024;	

  // 1) *********************************************  1)
	min_logLum = reduce_minmax(d_logLuminance, NUMTOLPIXEL,true);
	max_logLum = reduce_minmax(d_logLuminance, NUMTOLPIXEL,false);

  // ************************************************  2)
	const float lumRange = max_logLum - min_logLum;

  // ************************************************  3)
  // declare GPU memory
	int* d_bins;
  
  //allocate memory for it;
	checkCudaErrors( hipMalloc( (void **) & d_bins, sizeof(int) * numBins ));
  
  // initialize to 0 
    checkCudaErrors(hipMemset(d_bins, 0, sizeof(int) * numBins));  

  // Call histo kernel
	const int blockSize =  MAXTHREADPERBLOCK;
	int gridSize  = ceil(NUMTOLPIXEL / MAXTHREADPERBLOCK);
	histo<<<gridSize, blockSize>>>(d_bins, d_logLuminance, numRows, numCols, numBins, &min_logLum, &lumRange);
	
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  
  // 4)
  int acc = 0;
  for(int i = 0 ; i < numBins; ++i){
    d_cdf[i] = acc;
    acc += d_bins[i];
  }

  // Free memory
  checkCudaErrors(hipFree(d_bins));

}